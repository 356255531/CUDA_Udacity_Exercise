#include "hip/hip_runtime.h"
#include "utils.h"
#define LONG_RECTANGE 16
#define WIDE_RECTANGE 12
#define THREAD_PER_SM 192
#define SM_NUM 2

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage, 
                        unsigned char* const greyImage, 
                        int numRows, int numCols)
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    size_t j = blockDim.y * blockIdx.y + threadIdx.y;
    if ( i >= numRows || j >= numCols) return;

    uchar4 rgba = rgbaImage[i + j * numCols];
    unsigned char grey = static_cast<unsigned char>(rgba.x * .299f + rgba.y * .587f + rgba.z * .114f);
    greyImage[i + j * numCols] = grey;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
//You must fill in the correct sizes for the blockSize and gridSize
//currently only one block with one thread is being launched
    // size_t block_num_x = numRows / WIDE_RECTANGE + (numRows % WIDE_RECTANGE) == 0 ? 0 : 1;
    // size_t block_num_y = numCols / LONG_RECTANGE + (numCols % LONG_RECTANGE) == 0 ? 0 : 1;

    // const dim3 blockSize(block_num_x, block_num_y, 1);  //TODO
    // const dim3 gridSize( 1, 1, 1);  //TODO
    // rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
    // // *d_greyImage = *d_rgbaImage;
    // hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    int   blockWidth = 32;

    const dim3 blockSize(blockWidth, blockWidth, 1);
    int   blocksX = numRows/blockWidth+1;
    int   blocksY = numCols/blockWidth+1; //TODO
    const dim3 gridSize( blocksX, blocksY, 1);  //TODO
    rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);

    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
